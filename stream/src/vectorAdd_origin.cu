#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 /**
  * Vector addition: C = A + B.
  *
  * This sample is a very basic sample that implements element by element
  * vector addition. It is the same as the sample illustrating Chapter 2
  * of the programming guide with some additions like error checking.
  */

#include <stdio.h>

  // For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <omp.h>

#define START_GPU \
hipEvent_t     start, stop;\
float   elapsedTime;\
checkCudaErrors(hipEventCreate(&start)); \
checkCudaErrors(hipEventCreate(&stop));\
checkCudaErrors(hipEventRecord(start, 0));\

#define END_GPU \
checkCudaErrors(hipEventRecord(stop, 0));\
checkCudaErrors(hipEventSynchronize(stop));\
checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop)); \
printf("GPU Time used:  %3.1f ms\n", elapsedTime);\
checkCudaErrors(hipEventDestroy(start));\
checkCudaErrors(hipEventDestroy(stop));


#define START_CPU {\
double start = omp_get_wtime();

#define END_CPU \
double end = omp_get_wtime();\
double duration = end - start;\
printf("CPU Time used: %3.1f ms\n", duration * 1000);}


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const double* A, const double* B, double* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = cos(A[i]) / sin(B[i]);
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 100*1024*1024;
    size_t size = numElements * sizeof(double);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    double* h_A = (double*)malloc(size);

    // Allocate the host input vector B
    double* h_B = (double*)malloc(size);

    // Allocate the host output vector C
    double* h_C = (double*)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (double)RAND_MAX;
        h_B[i] = rand() / (double)RAND_MAX;
    }

    // Allocate the device input vector A
    START_GPU
    double* d_A = NULL;
    err = hipMalloc((void**)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
        printf("A ok!");

    // Allocate the device input vector B
    double* d_B = NULL;
    err = hipMalloc((void**)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
        printf("B ok!");

    // Allocate the device output vector C
    double* d_C = NULL;
    err = hipMalloc((void**)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
        printf("C ok!");

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
    

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    END_GPU

    START_CPU
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(cos(h_A[i])/sin( h_B[i]) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    END_CPU

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

