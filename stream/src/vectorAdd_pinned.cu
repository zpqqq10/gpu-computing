#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 /**
  * Vector addition: C = A + B.
  *
  * This sample is a very basic sample that implements element by element
  * vector addition. It is the same as the sample illustrating Chapter 2
  * of the programming guide with some additions like error checking.
  */

#include <stdio.h>

  // For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include "book.h"
#include <omp.h>

#define START_GPU \
hipEvent_t     start, stop;\
float   elapsedTime;\
checkCudaErrors(hipEventCreate(&start)); \
checkCudaErrors(hipEventCreate(&stop));\
checkCudaErrors(hipEventRecord(start, 0));\

#define END_GPU \
checkCudaErrors(hipEventRecord(stop, 0));\
checkCudaErrors(hipEventSynchronize(stop));\
checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop)); \
printf("GPU Time used:  %3.1f ms\n", elapsedTime);\
checkCudaErrors(hipEventDestroy(start));\
checkCudaErrors(hipEventDestroy(stop));


#define START_CPU {\
double start = omp_get_wtime();

#define END_CPU \
double end = omp_get_wtime();\
double duration = end - start;\
printf("CPU Time used: %3.1f ms\n", duration * 1000);}


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const double* A, const double* B, double* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = cos(A[i]) / sin(B[i]);
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    hipDeviceProp_t prop; 
    int whichDevice; 
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if(!prop.deviceOverlap){
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
        return 0;
    }
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 100*1024*1024;
    size_t size = numElements * sizeof(double);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    double* h_A = NULL;
    HANDLE_ERROR(hipHostAlloc((void**)&h_A, 
                        size, 
                        hipHostMallocDefault));

    // Allocate the host input vector B
    double* h_B = NULL;
    HANDLE_ERROR(hipHostAlloc((void**)&h_B, 
                        size, 
                        hipHostMallocDefault));

    // Allocate the host output vector C
    double* h_C = NULL;
    HANDLE_ERROR(hipHostAlloc((void**)&h_C, 
                        size, 
                        hipHostMallocDefault));

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (double)RAND_MAX;
        h_B[i] = rand() / (double)RAND_MAX;
    }

    // Allocate the device input vector A
    double* d_A = NULL;
    HANDLE_ERROR(hipMalloc((void**)&d_A, size));

    // Allocate the device input vector B
    double* d_B = NULL;
    HANDLE_ERROR(hipMalloc((void**)&d_B, size));

    // Allocate the device output vector C
    double* d_C = NULL;
    HANDLE_ERROR(hipMalloc((void**)&d_C, size));

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    START_GPU
    HANDLE_ERROR(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
    

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    HANDLE_ERROR(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    END_GPU

    START_CPU
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(cos(h_A[i])/sin( h_B[i]) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    END_CPU

    printf("Test PASSED\n");

    // Free device global memory
    HANDLE_ERROR(hipFree(d_A));

    HANDLE_ERROR(hipFree(d_B));

    HANDLE_ERROR(hipFree(d_C));

    // Free host memory
    HANDLE_ERROR(hipHostFree(h_A));
    HANDLE_ERROR(hipHostFree(h_B));
    HANDLE_ERROR(hipHostFree(h_C));
    // free(h_A);
    // free(h_B);
    // free(h_C);

    printf("Done\n");
    return 0;
}

